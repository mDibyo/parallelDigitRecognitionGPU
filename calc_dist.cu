#include <float.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include "utils.h"

float calc_min_dist(float *gpu_image, int i_width, int i_height,
										float* gpu_temp, int t_width) {

	float least_distance = UINT_MAX;

	if (t_width == 4096) {

		int threads_per_block = 512;
		int blocks_per_grid = 65564;

		int trans_height = i_height - t_width + 1;
		int trans_width = i_width - t_width + 1;
		int num_translations = trans_width * trans_height;

		float new_distance;

		size_t result_size = num_translations*sizeof(float);
		float* result = (float *)malloc(result_size);
		if (result == NULL) {
			printf("Unable to allocate space for result!\n");
			exit(EXIT_FAILURE);
		}
		float* gpu_result;
		CUDA_SAFE_CALL(hipMalloc(&gpu_result, result_size));

		size_t test_size = 100*sizeof(float);
		float* test = (float *)malloc(test_size);
		if (result == NULL) {
			printf("Unable to allocate space for result!\n");
			exit(EXIT_FAILURE);
		}
		float* gpu_test;
		CUDA_SAFE_CALL(hipMalloc(&gpu_result, test_size));

		// [16, 4096]
		dim3 dim_threads_per_block(threads_per_block, 1, 1);
		dim3 dim_blocks_per_grid(8, 4096);
		for (int i = 0; i < trans_height; i++) {
			for (int j = 0; j < trans_width; j++) {
				distance4096Kernel<<<dim_blocks_per_grid, dim_threads_per_block>>>
					(gpu_image, gpu_temp, i , j, i_width);
			}
		}

		CUDA_SAFE_CALL(hipFree(gpu_result));
		CUDA_SAFE_CALL(hipFree(gpu_test));

		free(result);
		free(test);

	}

	return least_distance;

}